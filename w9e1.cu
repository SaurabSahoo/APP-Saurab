
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void cos_sin_half(float *x, float *result, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N / 2) {
        result[idx] = cosf(x[idx]);
    } else if (idx < N) {
        result[idx] = sinf(x[idx]);
    }
}

__global__ void cos_sin_even_odd(float *x, float *result, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx % 2 == 0 && idx < N) {
        result[idx] = cosf(x[idx]);
    } else if (idx % 2 == 1 && idx < N) {
        result[idx] = sinf(x[idx]);
    }
}

int main() {
    int N = 1024; // Example size of vectors
    float *x, *result;
    float *d_x, *d_result;

    // Allocate memory on host
    x = (float*)malloc(N * sizeof(float));
    result = (float*)malloc(N * sizeof(float));

    // Initialize input vector
    for (int i = 0; i < N; i++) {
        x[i] = (float)i;
    }

    // Allocate memory on device
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_result, N * sizeof(float));

    // Copy input vector from host to device
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel for cos_sin_half
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    cos_sin_half<<<numBlocks, blockSize>>>(d_x, d_result, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cos_sin_half kernel execution time: %f milliseconds\n", milliseconds);

    // Copy result vector from device to host
    hipMemcpy(result, d_result, N * sizeof(float), hipMemcpyDeviceToHost);

    // Launch kernel for cos_sin_even_odd
    hipEventRecord(start);
    cos_sin_even_odd<<<numBlocks, blockSize>>>(d_x, d_result, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cos_sin_even_odd kernel execution time: %f milliseconds\n", milliseconds);

    // Copy result vector from device to host
    hipMemcpy(result, d_result, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
    hipFree(d_result);

    // Free host memory
    free(x);
    free(result);

    return 0;
}
